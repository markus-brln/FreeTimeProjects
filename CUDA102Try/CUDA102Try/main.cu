#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void vectorAdd(int* a, int n)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (tid < n)
		a[tid] = 2;

	//for (size_t idx = 0; idx < n; ++idx)
	//	a[idx] = 2;
}


int main()
{
    cout << "hello\n";

	int n = 256;
	size_t bytes = sizeof(int) * n;

	// Allocate host memory
	int *d_a;
	int *h_a = (int*)malloc(bytes);
	gpuErrchk(hipMalloc(&d_a, bytes));
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	// Threadblock size
	int NUM_THREADS = 256;

	// Grid size
	int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;

	// Launch kernel on default stream w/o shmem
	vectorAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_a, n);

	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

	for (size_t idx = 0; idx < n; ++idx)
		cout << h_a[idx];

    return 0;
}
