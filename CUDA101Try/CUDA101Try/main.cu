#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include "myclass/myclass.h"

__global__ void print_hello_global()
{
	printf("hello from device global function\n");
}



int main()
{
	print_hello_global << <2, 2 >> > ();

	hipDeviceSynchronize();

	MyClass myclass_obj;

	myclass_obj.call_global();

	return 0;
}