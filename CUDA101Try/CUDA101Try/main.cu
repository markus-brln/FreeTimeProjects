#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include ""
#include <iostream>
#include <vector>
#include <stdio.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}
// check for problems like too-old gpu driver version for your CUDA version
// from https://stackoverflow.com/questions/21990904/cuda-global-function-not-called


__device__ void hello_device()
{
	printf("hello from device-only function\n");
}


__global__ void call_device_function()
{
	hello_device();
}


__global__ void print_hello_device()
{
	printf("hello from device global function\n");
}

class GPU_Embassador
{
	int i;
	
	public:
		__device__ __host__ void assignValue(int val)
		{
			i = val;
		}

		__device__ __host__ int getValue()
		{
			return i;
		}
};

__global__ void manipulate_array(int* arr, size_t arr_size)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	// Calculate global thread ID

	// Boundary check
	if (tid < arr_size) 
		arr[tid] += 2;
}

__global__ void manipulate_class_obj(GPU_Embassador *obj)
{
	obj->assignValue(20);
}

void try_cudaMallocManaged()
{
	GPU_Embassador obj;
	obj.assignValue(10);
	GPU_Embassador* device_obj;
	hipMallocManaged(&device_obj, sizeof(GPU_Embassador));
	*device_obj = obj;
	manipulate_class_obj << <1, 1 >> > (device_obj);
	hipDeviceSynchronize();
	cout << "obj's data member changed? " << device_obj->getValue() << '\n';
}

void try_manipulate_std_vector()
{
	int vec_size = 1000;

	int NUM_THREADS = 256;							// Threadblock size
	int NUM_BLOCKS = (vec_size + NUM_THREADS - 1) / NUM_THREADS;	// Grid size

	vector<int> vec{ vec_size, 1 };
	cout << vec.size(); 

	int *host_dat = vec.data();
	int* dev_dat;
	int bytes = sizeof(int) * vec.size();
	hipMalloc(&dev_dat, bytes);
	hipMemcpy(dev_dat, host_dat, bytes, hipMemcpyHostToDevice);

	manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (dev_dat, vec.size());

	hipMemcpy(host_dat, dev_dat, bytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	cout << vec.size();
	//for (auto elem : vec)
		//cout << elem;
}


int main()
{
	try_manipulate_std_vector();

	exit(0);

	cout << "hello from host\n";
	print_hello_device << <1, 1 >> > ();
	call_device_function << <1, 1 >> > ();
	hipDeviceSynchronize();

	size_t n = 1 << 20;
	cout << "n: " << n << '\n';

	size_t bytes = sizeof(int) * n;
	int *h_a = (int*)malloc(bytes);					// Allocate host memory

	for (size_t idx = 0; idx < n; ++idx)
		h_a[idx] = 1;

	int* d_a;										// allocate device memory,
	gpuErrchk(hipMalloc(&d_a, bytes));				// check whether gpu driver version is ok etc
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

	
	int NUM_THREADS = 256;							// Threadblock size
	int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;	// Grid size

	for (size_t idx = 0; idx < 1000; ++idx)
	{
		manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (d_a, n);
		hipDeviceSynchronize();					// stop host code from doing stupid things
	}

	hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);


	for (size_t idx = 0; idx < 100; ++idx)			// check whether device actually did something
		cout << h_a[idx];



    return 0;
}
