#include "hip/hip_runtime.h"
#include "main.ih"

//int SIZE_X = 1366;
//int SIZE_Y = 768;
int SIZE_X = 768;
int SIZE_Y = 768;


__global__ void print_hello_device()
{
	printf("hello from global function\n");
}


int main()
{
	print_hello_device << <1, 1 >> > ();
	hipDeviceSynchronize();

	Model model;						// scene with objects
	sf::RenderWindow window{            // sf::Window is view 
		sf::VideoMode(SIZE_X, SIZE_Y),
		"Raytracing"
	};
	//window.setFramerateLimit(30);       // in case someone has a Threadripper CPU...

										// controller draws on window, changes model
	Controller controller{ model, window };

	controller.run();                   // start the game loop

    return 0;
}