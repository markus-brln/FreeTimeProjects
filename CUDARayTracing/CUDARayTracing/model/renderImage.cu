#include "hip/hip_runtime.h"
#include "model.h"


__global__ void manipulate_array(unsigned char* arr, size_t arr_size)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	// calculate global thread id

	// boundary check
	if (tid < arr_size)
	{
		arr[tid * 4] = 100;
		arr[tid * 4 + 3] = 255;
	}
}


unsigned char* Model::renderImage()
{
	int pixelNr = SIZE_X * SIZE_Y;

	/*for (int idx = 0; idx < pixelNr; ++idx)
	{
		d_pixels_host[idx * 4] = 255;
		d_pixels_host[idx * 4 + 3] = 255;
	}*/


	int NUM_THREADS = 1024;							// Threadblock size
	int NUM_BLOCKS = (pixelNr + NUM_THREADS - 1) / NUM_THREADS;	// Grid size


	unsigned char* host_dat = d_pixels_host.data();
	unsigned char* dev_dat;
	int bytes = sizeof(unsigned char) * pixelNr * 4;

	hipMalloc(&dev_dat, bytes);
	hipMemcpy(dev_dat, host_dat, bytes, hipMemcpyHostToDevice);

	manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (dev_dat, pixelNr);
	//::callFun<<<1, 1>>>(dev_dat, pixelNr, NUM_BLOCKS, NUM_THREADS);

	hipMemcpy(host_dat, dev_dat, bytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return host_dat;
}