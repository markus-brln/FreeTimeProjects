#include "hip/hip_runtime.h"
#include "model.h"
#include <vector>


__global__ void manipulate_array(unsigned char* arr, size_t arr_size)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	// calculate global thread id

	Color col{ 0.0, 0.0, 0.0 };


	// boundary check
	if (tid < arr_size)
	{
		arr[tid * 4] = 100;
		arr[tid * 4 + 3] = 255;
	}
}


unsigned char* Model::renderImage()
{
	int pixelNr = SIZE_X * SIZE_Y;

	int NUM_THREADS = 256;							// Threadblock size
	int NUM_BLOCKS = (pixelNr + NUM_THREADS - 1) / NUM_THREADS;	// Grid size


	unsigned char* host_dat = d_pixels_host.data();
	//unsigned char* dev_dat;
	int bytes = sizeof(unsigned char) * pixelNr * 4;

	//hipMalloc(&dev_dat, bytes);
	//hipMemcpy(dev_dat, host_dat, bytes, hipMemcpyHostToDevice);

	manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (d_pixels_device, pixelNr);
	//::callFun<<<1, 1>>>(dev_dat, pixelNr, NUM_BLOCKS, NUM_THREADS);

	hipMemcpy(host_dat, d_pixels_device, bytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return host_dat;
}