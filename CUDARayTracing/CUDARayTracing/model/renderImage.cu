#include "hip/hip_runtime.h"
#include "model.h"
#include "../raytracing/tracing.h"
#include <vector>


__global__ void manipulate_array(unsigned char* arr, size_t arr_size, Point screenCentre, 
	Point eye, Vector down, Vector right, size_t size_x, size_t size_y, Object **objects, size_t n_objects)
{
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	// calculate global thread id

	/*Color col{ 4.0, 4.0, 4.0 };
	Color col2{ 2, 2, 2 };
	Color col3 = col + col2;

	col3 = col3 * col2;

	col3 *= 3;
	col3.normalize();
	Triple t = col + col * 3;
	Triple t1 = t.reflect(col);

	Sphere s1{ Point{}, 20 };
	Hit h1 = Hit::NO_HIT();
	Ray r1 = Ray{ Point{0, 0, 0}, Vector{1, 0, 0} };*/
	//printf("%f ", t1.x);


	
	
	// 
	// boundary check
	if (tid < arr_size)
	{
		int x_coordinate = tid % size_x;
		int y_coordinate = tid / size_y;

		//if (y_coordinate < 50)
		//	printf("%d %d\n", x_coordinate, y_coordinate);
		// go from top left to bottom right, using the 3 rotated vectors
		Point pixel = screenCentre + right * (x_coordinate - size_x / 2) + down * (y_coordinate - size_y / 2);

		Ray ray(eye, (pixel - eye).normalized());   // shoot ray through pixel
		Color col = trace(ray, 1, objects, n_objects);					// recursion depth == 1
		//col.clamp();                                // some spots might be too bright
		arr[tid * 4] = 100;
		arr[tid * 4 + 3] = 255;
	}
}


unsigned char* Model::renderImage()
{
	int pixelNr = SIZE_X * SIZE_Y;

	int NUM_THREADS = 256;							// Threadblock size
	int NUM_BLOCKS = (pixelNr + NUM_THREADS - 1) / NUM_THREADS;	// Grid size
	int bytes = sizeof(unsigned char) * pixelNr * 4;	// n_bytes of image

	Vector down{ 0, -1, 0 };            // vector down from centre of screen
	Vector right{ 1, 0, 0 };            // vector right from centre

										// no Z-rotation yet
	rotateVector(down, d_eyeRotation.x, d_eyeRotation.y, 0);
	rotateVector(right, d_eyeRotation.x, d_eyeRotation.y, 0);

	// camera perpendicular to screen formed
	Vector d_camera = right.cross(down);// by right+down

	// rotate around the camera for Z-rotation (unfinished)
	// rotateVectorAroundVector(down, d_camera, eyeRotation.z);
	// rotateVectorAroundVector(right, d_camera, eyeRotation.z);

	Point screenCentre = d_eye + d_camera * SIZE_X * d_zoom;

	manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (d_pixelsDevice, pixelNr, screenCentre, 
		d_eye, down, right, SIZE_X, SIZE_Y, d_objectsDevice, n_objects);
	//::callFun<<<1, 1>>>(dev_dat, pixelNr, NUM_BLOCKS, NUM_THREADS);

	hipMemcpy(d_pixelsHost.data(), d_pixelsDevice, bytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return d_pixelsHost.data();
}