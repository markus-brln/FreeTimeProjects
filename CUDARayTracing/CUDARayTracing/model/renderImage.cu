#include "hip/hip_runtime.h"
#include "model.ih"
#include ""

//vector vec;



__global__ void callFun(unsigned char* dev_dat, size_t pixelNr, int NUM_BLOCKS, int NUM_THREADS)
{
	//int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Boundary check
	/*if (tid < pixelNr)
	{
		dev_dat[tid * 4] = 100;
		dev_dat[tid * 4 + 3] = 255;
	}*/
	//manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (dev_dat, pixelNr);
}


__global__ void manipulate_array(unsigned char* arr, size_t arr_size)
{
	//int tid = (blockIdx.x * blockDim.x) + threadIdx.x;	// Calculate global thread ID

	//// Boundary check
	//if (tid < arr_size)
	//{
	//	arr[tid * 4] = 100;
	//	arr[tid * 4 + 3] = 255;
	//}
}


unsigned char* Model::renderImage()
{
	int pixelNr = SIZE_X * SIZE_Y;

	/*for (int idx = 0; idx < pixelNr; ++idx)
	{
		d_pixels_host[idx * 4] = 255;
		d_pixels_host[idx * 4 + 3] = 255;
	}*/


	int NUM_THREADS = 256;							// Threadblock size
	int NUM_BLOCKS = (pixelNr + NUM_THREADS - 1) / NUM_THREADS;	// Grid size


	unsigned char *host_dat = d_pixels_host.data();
	unsigned char * dev_dat;
	int bytes = sizeof(unsigned char) * pixelNr * 4;

	hipMalloc(&dev_dat, bytes);
	hipMemcpy(dev_dat, host_dat, bytes, hipMemcpyHostToDevice);

	manipulate_array << <NUM_BLOCKS, NUM_THREADS >> > (dev_dat, pixelNr);
	//::callFun<<<1, 1>>>(dev_dat, pixelNr, NUM_BLOCKS, NUM_THREADS);

	hipMemcpy(host_dat, dev_dat, bytes, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return host_dat;
}